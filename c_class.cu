
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 250000

struct Strategy {
    double profitLoss;
    void (*backtest)(struct Strategy *, struct Tick *);
};

struct Tick {
    long timestamp;
    double open;
    double high;
    double low;
    double close;
    double rsi2;
    double rsi5;
    double rsi7;
    double rsi9;
    double rsi14;
    double stochastic5K;
    double stochastic5D;
    double stochastic10K;
    double stochastic10D;
    double stochastic14K;
    double stochastic14D;
};

__device__ void backtest(struct Strategy *self, struct Tick *tick) {
    int i;
    int j = 0;

    // Pretend to do something.
    // TODO: Actually do something useful.
    for (i=0; i<50; i++) {
        j++;
    }
}

__global__ void initializeStrategies(struct Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].profitLoss = 10000 + i;
        strategies[i].backtest = backtest;
    }
}

__global__ void backtestStrategies(struct Strategy *strategies, struct Tick *tick) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].backtest(&strategies[i], tick);
    }
}

int main() {
    int threadsPerBlock = 1000;
    int blockCount = N / threadsPerBlock;

    struct Strategy strategies[N];
    struct Strategy *devStrategies;
    int i;
    int j;

    int tickCount = 1000000;
    struct Tick *ticks = (Tick*) malloc(tickCount * sizeof(Tick));
    struct Tick *devTicks;
    int kFoldCount = 10;

    void (*backtester)(struct Strategy*, struct Tick*);

    backtester = &backtestStrategies;

    for (i=0; i<tickCount; i++) {
        ticks[i].timestamp = 1460611103;
        ticks[i].open = 89.5;
        ticks[i].high = 89.9;
        ticks[i].low = 89.2;
        ticks[i].close = 89.4;
        ticks[i].rsi2 = 89.7;
        ticks[i].rsi5 = 89.75;
        ticks[i].rsi7 = 89.72;
        ticks[i].rsi9 = 89.76;
        ticks[i].rsi14 = 89.9;
        ticks[i].stochastic5K = 89.2;
        ticks[i].stochastic5D = 89.4;
        ticks[i].stochastic10K = 89.7;
        ticks[i].stochastic10D = 89.75;
        ticks[i].stochastic14K = 89.72;
        ticks[i].stochastic14D = 89.76;
    }

    hipSetDevice(0);

    // Allocate memory on the GPU for the strategies.
    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));
    
    // Copy tick data to the GPU.
    hipMalloc((void**)&devTicks, N * sizeof(Tick));
    hipMemcpy(devTicks, ticks, N * sizeof(Tick), hipMemcpyHostToDevice);

    // Initialize strategies on the GPU.
    initializeStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);

    for (i=0; i<kFoldCount; i++) {
        for (j=0; j<tickCount; j++) {
            // Run backtests for all strategies.
            (*backtester)<<<blockCount, threadsPerBlock>>>(devStrategies, &devTicks[j]);
        }
    }

    // Free memory for the tick data from the GPU.
    hipFree(devTicks);

    // TODO: Determine if this is necessary.
    //cudaDeviceSynchronize();

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<N; i++) {
        printf("%f\n", strategies[i].profitLoss);
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}
