#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 200000

class Strategy {
    private:
        double profitLoss;

    public:
        __device__ __host__ Strategy(int initialProfitLoss) {
            this->profitLoss = initialProfitLoss;
        }
        __device__ __host__ void backtest() {
            int i = 0;
            int j = 0;

            for (i=0; i<50; i++) {
                j++;
            }

            this->profitLoss++;
        }
        __device__ __host__ double getProfitLoss() {
            return this->profitLoss;
        }
};

__global__ void backtestStrategies(Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].backtest();
    }
}

int main() {
    // int threadsPerBlock = 1000;
    // int blockCount = N / threadsPerBlock;
    int threadsPerBlock = 1024;
    int blockCount = 32;

    Strategy *devStrategies;
    Strategy *strategies = (Strategy*)malloc(N * sizeof(Strategy));
    double *data = (double*)malloc(1000 * sizeof(double));
    double *devData;
    int i = 0;

    hipSetDevice(0);

    // Allocate memory for strategies on the GPU.
    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));
    hipMalloc((void**)&devData, 1000 * sizeof(double));

    // Initialize strategies on host.
    for (i=0; i<N; i++) {
        strategies[i] = Strategy::Strategy(i);
    }

    // Copy strategies from host to GPU.
    hipMemcpy(devStrategies, strategies, N * sizeof(Strategy), hipMemcpyHostToDevice);

    for (i=0; i<363598; i++) {
        backtestStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);
        printf("\r%i", i);
    }

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);
    hipMemcpy(data, devData, 1000 * sizeof(double), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<1000; i++) {
        printf("%f\n", strategies[i].getProfitLoss());
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}