
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 200000

class Strategy {
    private:
        double profitLoss;

    public:
        __device__ __host__ Strategy() {
            this->profitLoss = 0;
        }
        __device__ __host__ void backtest() {
            this->profitLoss++;
        }
        __device__ __host__ double getProfitLoss() {
            return this->profitLoss;
        }
};

__global__ void backtestStrategies(Strategy *strategies) {
    // Reference: https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x)
    {
        strategies[i].backtest();
    }
}

int main() {
    int blockCount = 32;
    int threadsPerBlock = 1024;

    Strategy *devStrategies;
    Strategy *strategies = (Strategy*)malloc(N * sizeof(Strategy));
    int i = 0;

    // Allocate memory for strategies on the GPU.
    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));

    // Initialize strategies on host.
    for (i=0; i<N; i++) {
        strategies[i] = Strategy();
    }

    // Copy strategies from host to GPU.
    hipMemcpy(devStrategies, strategies, N * sizeof(Strategy), hipMemcpyHostToDevice);

    for (i=0; i<363598; i++) {
        backtestStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);
    }

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<N; i++) {
        printf("%f\n", strategies[i].getProfitLoss());
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}