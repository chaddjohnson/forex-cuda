#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 200000

class Strategy {
    private:
        double profitLoss;

    public:
        __device__ Strategy() {
            this->profitLoss = 0;
        }
        __device__ void backtest() {
            int i = 0;
            int j = 0;

            for (i=0; i<50; i++) {
                j++;
            }

            this->profitLoss += 1.0;
        }
        __device__ __host__ double getProfitLoss() {
            return this->profitLoss;
        }
};

__global__ void initializeStrategies(Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i] = Strategy::Strategy();
    }
}

__global__ void backtestStrategies(Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].backtest();
    }
}

int main() {
    // int threadsPerBlock = 1000;
    // int blockCount = N / threadsPerBlock;
    int threadsPerBlock = 1024;
    int blockCount = 32;

    Strategy *devStrategies;
    Strategy *strategies = (Strategy*)malloc(N * sizeof(Strategy));
    int i = 0;

    hipSetDevice(0);

    // Allocate memory for strategies on the GPU.
    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));

    // Initialize strategies on the GPU.
    initializeStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);

    for (i=0; i<3635988; i++) {
        backtestStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);
    }

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<1000; i++) {
        printf("%f\n", strategies[i].getProfitLoss());
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}