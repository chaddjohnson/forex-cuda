
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CONFIGURATION_COUNT 250000

struct Tick {
    long timestamp;
    double open;
    double high;
    double low;
    double close;
    double sma13;
    double ema50;
    double ema100;
    double ema200;
    double rsi;
    double stochK;
    double stochD;
    double prcUpper;
    double prcLower;
};

struct Strategy {
    double profitLoss;
    void (*backtest)(Strategy *, Tick *);
};

__device__ void backtest(Strategy *self, Tick *tick) {
    int i;
    int j = 0;

    // Pretend to do something.
    // TODO: Actually do something useful.
    for (i=0; i<50; i++) {
        j++;
    }
}

__global__ void initializeStrategies(Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < CONFIGURATION_COUNT) {
        strategies[i].profitLoss = 10000 + i;
        strategies[i].backtest = backtest;
    }
}

__global__ void backtestStrategies(Strategy *strategies, Tick *tick) {
    // TODO: Harness multiple dimensions?
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < CONFIGURATION_COUNT) {
        strategies[i].backtest(&strategies[i], tick);
    }
}

int main() {
    int threadsPerBlock = 1000;
    int blockCount = CONFIGURATION_COUNT / threadsPerBlock;

    Strategy strategies[CONFIGURATION_COUNT];
    Strategy *devStrategies;
    int i = 0;
    int j = 0;
    int k = 0;

    int tickCount = 1000000;
    Tick *ticks = (Tick*) malloc(CONFIGURATION_COUNT * sizeof(Tick));;
    Tick *devTicks;
    int kFoldCount = 10;

    void (*backtester)(Strategy*, Tick*);
    backtester = &backtestStrategies;

    hipSetDevice(0);

    // Allocate memory on the GPU for the strategies.
    hipMalloc((void**)&devStrategies, CONFIGURATION_COUNT * sizeof(Strategy));

    // Allocate memory on the GPU for the ticks.
    hipMalloc((void**)&devTicks, CONFIGURATION_COUNT * sizeof(Tick));

    // Initialize strategies on the GPU.
    initializeStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);

    // Run through each k-fold step.
    for (i=0; i<kFoldCount; i++) {
        // Run through every tick.
        for (j=0; j<tickCount; j++) {
            printf("%i\n", j);

            if (j > 0) {
                // Wait for currently-running kernels to finish.
                hipDeviceSynchronize();

                // Free currently-allocated GPU memory, and allocate more.
                hipFree(devTicks);

                // Clear host memory for previous ticks.
                memset(ticks, 0, CONFIGURATION_COUNT * sizeof(Tick));
            }

            // Set up data for every configuration.
            for (k=0; k<CONFIGURATION_COUNT; k++) {
                ticks[k].timestamp = 1460611103;
                ticks[k].open = 89.5;
                ticks[k].high = 89.5;
                ticks[k].low = 89.5;
                ticks[k].close = 89.5;
                ticks[k].sma13 = 89.5;
                ticks[k].ema50 = 89.5;
                ticks[k].ema100 = 89.5;
                ticks[k].ema200 = 89.5;
                ticks[k].rsi = 89.5;
                ticks[k].stochK = 89.5;
                ticks[k].stochD = 89.5;
                ticks[k].prcUpper = 89.5;
                ticks[k].prcLower = 89.5;
            }

            // Copy ticks to the GPU.
            hipMemcpy(devTicks, ticks, CONFIGURATION_COUNT * sizeof(Tick), hipMemcpyHostToDevice);

            // Run backtests for all strategy configurations.
            (*backtester)<<<blockCount, threadsPerBlock>>>(devStrategies, devTicks);
        }
    }

    // Free memory for the tick data from the GPU.
    hipFree(devTicks);

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, CONFIGURATION_COUNT * sizeof(Strategy), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<CONFIGURATION_COUNT; i++) {
        printf("%f\n", strategies[i].profitLoss);
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}
