#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/device_ptr.h>
#include <thrust/device_new.h>

#define N 512

using namespace thrust;

class Study {
    public:
        int data[N];

        __device__ __host__ Study() {
            calculate();
        }

        __device__ __host__ void calculate() {
            for (int i=0; i<N; i++) {
                data[i] = i*2;
            }
        }
};

__global__ void test(Study* s) {
    for (int i=0; i<N; i++)
        printf("%d\n", s->data[i]);
}

int main() {
    device_ptr<Study> s = device_new<Study>();
    test<<<1,1>>>(thrust::raw_pointer_cast(s));

    hipDeviceSynchronize();
    printf("Done!\n");
}
