#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/device_ptr.h>
#include <thrust/device_new.h>

#define N 512

class Study {
    public:
        int data[N];

        __device__ __host__ Study() {
            calculate();
        }

        __device__ __host__ void calculate() {
            for (int i=0; i<N; i++) {
                data[i] = i*2;
            }
        }
};

__global__ void test(Study* s) {
    for (int i=0; i<N; i++)
        printf("%d\n", s->data[i]);
}

int main() {
    thrust::device_ptr<Study> s = thrust::device_new<Study>();
    test<<<1,1>>>(thrust::raw_pointer_cast(s));

    hipDeviceSynchronize();
    printf("Done!\n");
}
