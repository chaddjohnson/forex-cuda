#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/device_ptr.h>
#include <thrust/device_new.h>

#define N 1024000

__global__ void add(int *data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        data[i]++;
    }
}

int main() {
    int data[N];
    int *dev_data;
    int i;

    // Allocate memory on the GPU.
    hipMalloc((void**)&dev_data, N * sizeof(int));

    // Initialize data.
    for (i=0; i<N; i++) {
        data[i] = 0;
    }

    // Copy data to the GPU.
    hipMemcpy(dev_data, data, N * sizeof(int), hipMemcpyHostToDevice);

    for (i=0; i<100; i++) {
        add<<<1000, 1024>>>(dev_data);
    }

    hipDeviceSynchronize();

    // Copy data from the GPU.
    hipMemcpy(data, dev_data, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free memory allocated on the GPU.
    hipFree(dev_data);

    // for (i=0; i<N; i++) {
    //     printf("%d\n", data[i]);
    // }

    return 0;
}
