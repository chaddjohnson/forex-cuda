
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 250000

struct Strategy {
    double profitLoss;
    void (*backtest)(struct Strategy *);
};

__device__ void backtest(struct Strategy *self) {
    self->profitLoss++;
}

__global__ void runBacktest(struct Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].backtest(&strategies[i]);
    }
}

__global__ void initializeStrategies(struct Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].profitLoss = 10000 + i;
        strategies[i].backtest = backtest;
    }
}

int main() {
    int threadsPerBlock = 10;
    int blockCount = N / threadsPerBlock;

    struct Strategy strategies[N];
    struct Strategy *devStrategies;
    int i;

    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));
    hipMemcpy(devStrategies, strategies, N * sizeof(Strategy), hipMemcpyHostToDevice);

    initializeStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);
    runBacktest<<<blockCount, threadsPerBlock>>>(devStrategies);

    hipDeviceSynchronize();
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);

    for (i=0; i<N; i++) {
        printf("%f\n", strategies[i].profitLoss);
    }

    hipFree(devStrategies);

    return 0;
}
