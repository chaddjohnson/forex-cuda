
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 250000

struct Strategy {
    double profitLoss;
    void (*backtest)(struct Strategy *, struct Tick *);
};

struct Tick {
    long timestamp;
    double open;
    double high;
    double low;
    double close;
    double rsi2;
    double rsi5;
    double rsi7;
    double rsi9;
    double rsi14;
    double stochastic5K;
    double stochastic5D;
    double stochastic10K;
    double stochastic10D;
    double stochastic14K;
    double stochastic14D;
};

__device__ void backtest(struct Strategy *self, struct Tick *tick) {
    int i;
    int j = 0;

    // Pretend to do something.
    // TODO: Actually do something useful.
    for (i=0; i<50; i++) {
        j++;
    }
}

__global__ void initializeStrategies(struct Strategy *strategies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].profitLoss = 10000 + i;
        strategies[i].backtest = backtest;
    }
}

__global__ void backtestStrategies(struct Strategy *strategies, struct Tick *tick) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        strategies[i].backtest(&strategies[i], tick);
    }
}

int main() {
    int threadsPerBlock = 1000;
    int blockCount = N / threadsPerBlock;

    struct Strategy strategies[N];
    struct Strategy *devStrategies;
    int i;

    struct Tick tick = {1460611103, 89.5, 89.9, 89.2, 89.4, 89.7, 89.75, 89.72, 89.76, 89.9, 89.2, 89.4, 89.7, 89.75, 89.72, 89.76};
    struct Tick *devTick;

    hipSetDevice(0);

    // Allocate memory on the GPU for the strategies.
    // TODO: Allocate memory on all GPUs.
    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));
    
    // Initialize strategies on the GPU.
    initializeStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);

    for (i=0; i<1000000; i++) {
        // Copy tick data to the GPU.
        // TODO: Copy to all GPUs.
        hipMalloc((void**)&devTick, sizeof(Tick));
        hipMemcpy(devTick, &tick, sizeof(Tick), hipMemcpyHostToDevice);

        // Run backtests for all strategies.
        // TODO: Run on all GPUs.
        backtestStrategies<<<blockCount, threadsPerBlock>>>(devStrategies, devTick);

        // Free memory for the tick from the GPU.
        hipFree(devTick);
    }

    // TODO: Determine if this is necessary.
    //cudaDeviceSynchronize();

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<N; i++) {
        printf("%f\n", strategies[i].profitLoss);
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}
